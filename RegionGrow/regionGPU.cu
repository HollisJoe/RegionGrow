#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include"tgt\vector.h"
#include<hip/hip_runtime_api.h>
#include<vector>

#include <stdio.h>
#include<time.h>


__global__ void kernel(unsigned char* vol, unsigned char* seedsVol, int3 volDim, size_t* numSeeds)
{
	int3 pos;
	pos.x = blockDim.x*blockIdx.x + threadIdx.x;
	pos.y = blockDim.y*blockIdx.y + threadIdx.y;
	pos.z = blockDim.z*blockIdx.z + threadIdx.z;

	if (pos.x > volDim.x /64)
	{
		return;
	}

	if (pos.y > volDim.y - 2 || pos.y <1)
	{
		return;
	}

	if (pos.z > volDim.z - 2 || pos.z <1)
	{
		return;
	}

	//ÿ���̴߳��� x ������������64������
	for (size_t i = 0; i < 64; i++)
	{
		size_t index = pos.z*volDim.x*volDim.y + pos.y*volDim.x + pos.x + i;

		if (vol[index] == unsigned char(0) || seedsVol[index] == unsigned char(255))
		{
			//���ĳ�ε���,�����̶߳�����һ��·��,˵����������

			return;
		}
		else
		{
			//�ж��������� 6 ����

			size_t x_r = pos.z*volDim.x*volDim.y + pos.y*volDim.x + pos.x + 1 + i;
			size_t x_l = pos.z*volDim.x*volDim.y + pos.y*volDim.x + pos.x - 1 + i;
			size_t y_r = pos.z*volDim.x*volDim.y + (pos.y + 1)*volDim.x + pos.x + i;
			size_t y_l = pos.z*volDim.x*volDim.y + (pos.y - 1)*volDim.x + pos.x + i;
			size_t z_r = (pos.z + 1)*volDim.x*volDim.y + pos.y*volDim.x + pos.x + i;
			size_t z_l = (pos.z - 1)*volDim.x*volDim.y + pos.y*volDim.x + pos.x + i;

			if (seedsVol[x_r] | seedsVol[x_l] | seedsVol[y_r] | seedsVol[y_l] | seedsVol[z_r] | seedsVol[z_l])
			{
				atomicAdd(numSeeds, 1);
				seedsVol[index] = unsigned char(255);
			}
		}
	}

	

}

extern "C"
void RegionGrowGPU(unsigned char* inVol, unsigned char* outVol, tgt::svec3 volDim, std::vector<tgt::svec3>& seeds)
{
	size_t numVoxels = volDim.x*volDim.y*volDim.z;

	unsigned char* d_volume,*d_seedVolume;
	checkCudaErrors(hipMalloc((void **)&d_volume, numVoxels*sizeof(unsigned char)));
	checkCudaErrors(hipMalloc((void **)&d_seedVolume, numVoxels*sizeof(unsigned char)));

	hipError_t err;

	err = hipMemcpy(d_volume, inVol, numVoxels*sizeof(unsigned char), hipMemcpyHostToDevice);
	
	unsigned char* seedVol = new unsigned char[numVoxels];
	memset(seedVol, unsigned char(0), numVoxels*sizeof(unsigned char));

	for (size_t i = 0; i < seeds.size(); i++)
	{
		tgt::svec3 pos = seeds.at(i);
		seedVol[pos.z*volDim.x*volDim.y + pos.y*volDim.x + pos.x] = unsigned char(255);
	}

	err = hipMemcpy(d_seedVolume, seedVol, numVoxels*sizeof(unsigned char), hipMemcpyHostToDevice);

	delete[] seedVol;
	seedVol = nullptr;

	int3 vd = make_int3(volDim.x, volDim.y, volDim.z);

	dim3 dimGrid((volDim.x + 7) / 64, (volDim.y + 7) / 8, (volDim.z + 7) / 8);
	dim3 dimBlock(1, 8, 8);

	/*for (size_t i = 0; i < 500; i++)
	{
		kernel << <dimGrid, dimBlock >> >(d_volume, d_seedVolume, vd);
	}*/

	size_t *d_numSeeds;
	size_t numSeeds = 0, lastValue = 0;


	hipMalloc((void**)&d_numSeeds, sizeof(size_t));

	hipMemcpy(d_numSeeds, &numSeeds, sizeof(size_t), hipMemcpyHostToDevice);


	int i = 0;

	while (true)
	{
		i++;

		kernel << <dimGrid, dimBlock >> >(d_volume, d_seedVolume, vd,d_numSeeds);

		//�ж����ӵ��Ƿ�����
		hipMemcpy(&numSeeds, d_numSeeds, sizeof(size_t), hipMemcpyDeviceToHost);

		if (i>400)
		{
			break;
		}

		//if (numSeeds == lastValue)
		//{
		//	std::cout << "i " << i << std::endl;
		//	break;
		//}
		//else
		//{
		//	lastValue = numSeeds;
		//}
	}

	checkCudaErrors(hipPeekAtLastError());
	checkCudaErrors(hipDeviceSynchronize());

	err = hipMemcpy(outVol, d_seedVolume, numVoxels*sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(d_volume);
	hipFree(d_seedVolume);

}